
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void hadamardProduct (int *a, int *b, int *c, int N);


int main()
{
	int N = 32;
	int BLOCK_DIM = 32;
	int size = N * N * sizeof(int);
	int *a = (int *)calloc(N*N, sizeof(int));
	int *b = (int *)calloc(N*N, sizeof(int));
	int *c = (int *)calloc(N*N, sizeof(int));
	int *dev_a = NULL, *dev_b  = NULL, *dev_c  = NULL;

	for(int i = 0; i < N*N; i++)
	{
	    a[i] = 2;
	    b[i] = 7;
	}

	// initialize a and b with real values (NOT SHOWN)
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	//gettimeofday(&start, NULL);

	hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

	dim3 threads(BLOCK_DIM, BLOCK_DIM);
	dim3 grid((int)ceil(N/threads.x),(int)ceil(N/threads.y));

	hadamardProduct<<<grid,threads>>>(dev_a,dev_b,dev_c, N);
	hipDeviceSynchronize();

	hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++)
	{
	    for(int j = 0; j < N; j++)
	    {
		printf("%d ", c[i*j]);
	    }
	    printf("\n");
	}
	//gettimeofday(&end, NULL);
	printf("GPU for %i additions\n", N);


	hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
}


__global__ void hadamardProduct (int *a, int *b, int *c, int N)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = col + row * N;
	if (col < N && row < N)
	{
		c[index] = a[index] * b[index];
	}
}
