// Compile using nvcc <file> -lcublas -o <output>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <sstream>
#include <iostream>

// Define block size for thread allocation
#define NUM_THREADS 32 // 32 is max for N^2 threads: 32*32 = 1024
#define LOGGING 0

//==================================================================================
//=== Structure definitions
//==================================================================================
typedef struct _kernelParams {
    int block_size;
    int grid_size;
} sKernelParams;

typedef struct _matrixSize // Optional Command-line multiplier for matrix sizes
{
    unsigned int A_height, A_width, B_height, B_width, C_height, C_width;
} MatrixSize;

typedef struct _vSize // Optional Command-line multiplier for matrix sizes
{
    unsigned int len_A, len_B, len_C;
} VectorSize;

//==================================================================================
//=== Structure functions
//==================================================================================

/**
 * @brief -  sets values of vector size structure
 *
 * @param vector_size - pointer to vector size struct
 * @param len - length of all vectors
 */
void SetVectorSize(VectorSize *vector_size, unsigned int len) {
    vector_size->len_A = len;
    vector_size->len_B = len;
    vector_size->len_C = len;

    if (LOGGING == 1)
        fprintf(stdout, "Vector A(%u), Vector B(%u), Vector C(%u)\n",
                vector_size->len_A,
                vector_size->len_B,
                vector_size->len_C);

    if (vector_size->len_A != vector_size->len_B ||
        vector_size->len_B != vector_size->len_C ||
        vector_size->len_C != vector_size->len_A) {
        fprintf(stderr, "ERROR: Vector lengths do not match!\n");
        exit(-1);
    }
}

/**
 * @brief -  sets values of matrix size structure
 *
 * @param matrixSize - reference to matrix size struct
 * @param widthA - width of matrix A
 * @param heightA - height of matrix A
 * @param widthB - width of matrix B
 * @param heightB - height of matrix B
 * @param widthC - width of matrix C
 * @param heightC - height of matrix C
 */
void SetMatrixSize(MatrixSize *matrixSize,
                   unsigned int widthA, unsigned int heightA,
                   unsigned int widthB, unsigned int heightB,
                   unsigned int widthC, unsigned int heightC) {
    matrixSize->A_height = heightA;
    matrixSize->A_width = widthA;
    matrixSize->B_height = heightB;
    matrixSize->B_width = widthB;
    matrixSize->C_height = heightC;
    matrixSize->C_width = widthC;

    if (LOGGING == 1)
        fprintf(stdout, "Matrix A(%u x %u), Matrix B(%u x %u), Matrix C(%u x %u)\n",
                matrixSize->A_width,
                matrixSize->A_height,
                matrixSize->B_width,
                matrixSize->B_height,
                matrixSize->C_width,
                matrixSize->C_height);

}

//==================================================================================
//=== GPU memory initialization functions
//==================================================================================

/**
 * @brief - allocates memory on GPU for vectors A, B, and C then copies the values for vector A and B
 *          from host PC onto the device
 *
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param vector_size - reference to vector size structure
 * @param host_vA - pointer to host vector A (with values)
 * @param host_vB - pointer to host vector B (with values)
 * @param dev_A - pointer to vector A device memory reference
 * @param dev_B - pointer to vector B device memory reference
 * @param dev_C - pointer to vector C device memory reference
 */
void VectorInitCUDA(int argc, char **argv, int devID, VectorSize *vector_size, float *host_vA, float *host_vB,
                    float *&dev_A, float *&dev_B, float *&dev_C) {
    // Assign CUDA variables
    hipError_t err;

    // Assign size variables
    size_t size_A = vector_size->len_A * sizeof(float);
    size_t size_B = vector_size->len_B * sizeof(float);
    size_t size_C = vector_size->len_C * sizeof(float);

    // Allocate memory on GPU
    err = hipMalloc((void **) &dev_A, size_A);
    if (err != hipSuccess) fprintf(stderr, "ERROR allocating vector A: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_B, size_B);
    if (err != hipSuccess) fprintf(stderr, "ERROR allocating vector B: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_C, size_C);
    if (err != hipSuccess) fprintf(stderr, "ERROR allocating vector C: %s\n", hipGetErrorString(err));

    // Copy data from host PC to GPU
    err = hipMemcpy(dev_A, host_vA, size_A, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying vector A to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_B, host_vB, size_B, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying vector B to GPU: %s\n", hipGetErrorString(err));

}

/**
 * @brief - allocates memory on GPU for matrices A, B, and C then copies the values for matrices A, B and C
 *          from host PC onto the device
 *
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to vector size structure
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param dev_matrixA - pointer to matrix A device memory reference
 * @param dev_matrixB - pointer to matrix B device memory reference
 * @param dev_matrixC - pointer to matrix C device memory reference
 */
void MatrixInitCUDA(int argc, char **argv, int &devID, MatrixSize *matrixSize,
                    float *host_matrixA, float *host_matrixB, float *host_matrixC,
                    float *&dev_matrixA, float *&dev_matrixB, float *&dev_matrixC) {
    // Assign CUDA variables
    hipError_t err;

    // Assign size variables
    size_t matrixA_size = matrixSize->A_height * matrixSize->A_width * sizeof(float);
    size_t matrixB_size = matrixSize->B_height * matrixSize->B_width * sizeof(float);
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Allocate memory on GPU
    err = hipMalloc((void **) &dev_matrixA, matrixA_size);
    if (err != hipSuccess) fprintf(stderr, "ERROR allocating matrix A: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_matrixB, matrixB_size);
    if (err != hipSuccess) fprintf(stderr, "ERROR allocating matrix B: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_matrixC, matrixC_size);
    if (err != hipSuccess) fprintf(stderr, "ERROR allocating matrix C: %s\n", hipGetErrorString(err));

    // Copy data from host PC to GPU
    err = hipMemcpy(dev_matrixA, host_matrixA, matrixA_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying matrix A to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_matrixB, host_matrixB, matrixB_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying matrix B to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_matrixC, host_matrixC, matrixC_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying matrix C to GPU: %s\n", hipGetErrorString(err));
}

//==================================================================================
//=== CUDA Vector Kernels
//==================================================================================
/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of a vector addition
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param dev_vecC - pointer to device memory for vector C
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorAdditionKernel(float *dev_vecA, float *dev_vecB, float *dev_vecC,
                                     float alpha, float beta, int vecLen) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen) {
        dev_vecC[i] = alpha * dev_vecA[i] + beta * dev_vecB[i];
    }
}

/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of a vector hadamard product
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param dev_vecC - pointer to device memory for vector C
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorHadamardKernel(float *dev_vecA, float *dev_vecB, float *dev_vecC,
                                     float alpha, float beta, int vecLen) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen) {
        dev_vecC[i] = alpha * dev_vecA[i] * beta * dev_vecB[i];
    }
}

/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 *           REMEMBER: Call kernel using: <<<grid, threads, vecLen>>>
 * @brief - kernel for GPU computation of a vector dot product
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param result - pointer to a single float value where the result will be returned
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorDotProduct(float *dev_vecA, float *dev_vecB, float *result,
                                 float alpha, float beta, int vecLen) {
    extern __shared__ float temp[];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen) {
        temp[i] = alpha * dev_vecA[i] * beta * dev_vecB[i];
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        float sum = 0.0;
        for (int j = 0; j < vecLen; j++) {
            sum += temp[j];
        }
        result[0] = sum;
    }
}

/**
 * @required INPUT AND OUTPUT VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of the vector sigmoid function
 * @param dev_matrixA - pointer to device memory for vector A
 * @param dev_matrixC - pointer to device memory for vector C
 * @param vecLen - length of all vectors
 */
__global__ void VectorSigmoid(float *dev_vecA, float *dev_vecC, int vecLen) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < vecLen) {
        float exp = 1 + expf(-dev_vecA[index]);
        dev_vecC[index] = 1 / exp;
    }
}

/**
 * @required INPUT AND OUTPUT VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of the vector sigmoid derivative function
 * @param dev_matrixA - pointer to device memory for vector A
 * @param dev_matrixC - pointer to device memory for vector C
 * @param vecLen - length of all vectors
 */
__global__ void VectorSigmoidDerivative(float *dev_vecA, float *dev_vecC, int vecLen) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < vecLen) {
        float exp = 1 + expf(-dev_vecA[index]);
        float sig = 1 / exp;
        dev_vecC[index] = sig * (1 - sig);
    }
}

//==================================================================================
//=== CUDA Vector Kernel Drivers
//==================================================================================

/**
 * @brief driver function for computing vector operations
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param vectorSize - reference to vector size structure
 * @param operation - switch-case value for which matrix operation to perform
 *                    1: Vector addition
 *                    2: Vector Hadamard product
 *                    3: Vector dot product
 *                    4: Vector sigmoid function
 *                    5: Vector sigmoid derivative
 * @param host_vectorA - pointer to host vector A (with values)
 * @param host_vectorB - pointer to host vector B (with values)
 * @param host_vectorC - pointer to host vector C (with values)
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 */
void RunVectorKernel(int argc, char **argv, int &devID, VectorSize *vectorSize, int operation,
                     float *host_vectorA, float *host_vectorB, float *host_vectorC, float alpha, float beta) {
    // Assign CUDA variables
    hipError_t err;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    int gridX = (int) ceil((float) vectorSize->len_C / (float) threads.x);
    int gridY = (int) ceil((float) vectorSize->len_C / (float) threads.y);
    dim3 grid((unsigned int) gridX, (unsigned int) gridY);

    // Assign computation variables
    float *dev_vectorA = NULL;
    float *dev_vectorB = NULL;
    float *dev_vectorC = NULL;

    size_t vectorC_size = vectorSize->len_C * sizeof(float);

    // Initialize memory on GPU
    VectorInitCUDA(argc, argv, devID, vectorSize, host_vectorA, host_vectorB, dev_vectorA, dev_vectorB, dev_vectorC);

    switch (operation) {
        case 1: {
            // Compute vector addition
            VectorAdditionKernel<<<grid, threads>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta,
                    vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) fprintf(stderr, "ERROR in Vector Add Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 2: {
            // Compute vector Hadamard Product
            VectorHadamardKernel<<<grid, threads>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta,
                    vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Vector Hadamard Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 3: {
            // Compute vector dot product
            VectorDotProduct<<<grid, threads, vectorSize->len_C>>>
                                              (dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Vector Dot product Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 4: {
            // Compute sigmoid function
            VectorSigmoid<<<grid, threads>>>(dev_vectorA, dev_vectorC, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Vector Sigmoid Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 5: {
            // Compute sigmoid derivative
            VectorSigmoidDerivative<<<grid, threads>>>(dev_vectorA, dev_vectorC, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Vector Sigmoid Derivative Computation: %s\n", hipGetErrorString(err));
            break;
        }
        default: {
            fprintf(stderr, "ERROR: No vector kernel selected. Operation Aborted");
            break;
        }
    }

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        fprintf(stderr, "ERROR synchronizing Vector Kernel calculation: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_vectorC, dev_vectorC, vectorC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        fprintf(stderr, "ERROR copying vector C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_vectorA);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing vector A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_vectorB);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing vector B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_vectorC);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing vector C on GPU: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) fprintf(stderr, "ERROR synchronizing Vector Kernel End: %s\n", hipGetErrorString(err));
    if (LOGGING == 1) fprintf(stdout, "Vector Kernel finished.\n");
}

//==================================================================================
//=== CUDA Matrix Kernels
//==================================================================================

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of matrix additions
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixB - pointer to device memory for matrix B
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixAddKernel(float *dev_matrixA, float *dev_matrixB, float *dev_matrixC,
                                float alpha, float beta, int matrix_width, int matrix_height) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height) {
        dev_matrixC[index] = alpha * dev_matrixA[index] + beta * dev_matrixB[index];
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for actual GPU computation for the matrix Hadamard product
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixB - pointer to device memory for matrix B
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixHadamardKernel(float *dev_matrixA, float *dev_matrixB, float *dev_matrixC,
                                     float alpha, float beta, int matrix_width, int matrix_height) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height) {
        dev_matrixC[index] = alpha * dev_matrixA[index] * beta * dev_matrixB[index];
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of matrix sigmoid function
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixSigmoid(float *dev_matrixA, float *dev_matrixC,
                              int matrix_width, int matrix_height) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height) {
        float exp = 1 + expf(-dev_matrixA[index]);
        dev_matrixC[index] = 1 / exp;
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of the matrix sigmoid derivative function
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixSigmoidDerivative(float *dev_matrixA, float *dev_matrixC,
                                        int matrix_width, int matrix_height) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height) {
        float exp = 1 + expf(-dev_matrixA[index]);
        float sig = 1 / exp;
        dev_matrixC[index] = sig * (1 - sig);
    }
}

//==================================================================================
//=== CUDA Matrix Kernel Drivers
//==================================================================================

/**
 * @brief -  Uses CUBLAS library to perform alpha(A x B) + beta(C) matrix multiplication and addition
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to vector size structure
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param alpha - value for alpha in CUBLAS function
 * @param beta - value for beta in CUBLAS function
 * @param transposeA - true if A should be transposed
 * @param transposeB - true if B should be transposed
 */

void MatrixMultiplyCUBLAS(int argc, char **argv, int &devID, MatrixSize *matrixSize,
                          float *host_matrixA, float *host_matrixB, float *host_matrixC,
                          float alpha, float beta, bool transposeA, bool transposeB) {
    // Assign CUDA variables
    hipblasHandle_t handle;
    hipError_t err;
    hipblasCreate(&handle);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devID);
    dim3 threads(NUM_THREADS, NUM_THREADS);
    dim3 grid(matrixSize->C_width / threads.x, matrixSize->C_height / threads.y);

    // Assign computation variables
    float *dev_matrixA = NULL, *dev_matrixB = NULL, *dev_matrixC = NULL;
    int m = matrixSize->A_height;
    int n = matrixSize->B_width;
    int k = matrixSize->A_width;
    hipblasOperation_t transA = HIPBLAS_OP_N, transB = HIPBLAS_OP_N;
    if (transposeA) transA = HIPBLAS_OP_T;
    if (transposeB) transB = HIPBLAS_OP_T;
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Initialize memory on GPU
    MatrixInitCUDA(argc, argv, devID, matrixSize,
                   host_matrixA, host_matrixB, host_matrixC,
                   dev_matrixA, dev_matrixB, dev_matrixC);

    // Perform matrix multiplication
    // SGEMM PARAMS: (handle, transposeA, transposeB, m, n, k, alpha, matrix A, k, matrix B, n, beta, matrix C, n)
    hipblasSgemm(handle, transA, transB, m, n, k, &alpha, dev_matrixA, m,
                dev_matrixB, n, &beta, dev_matrixC, m);
    err = hipGetLastError();
    if (err != hipSuccess) fprintf(stderr, "ERROR in SGEMM: %s\n", hipGetErrorString(err));

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) fprintf(stderr, "ERROR synchronizing SGEMM calculation: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_matrixC, dev_matrixC, matrixC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying matrix C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_matrixA);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing matrix A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixB);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing matrix B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixC);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing matrix C on GPU: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) fprintf(stderr, "ERROR synchronizing SGEMM end: %s\n", hipGetErrorString(err));
    if (LOGGING == 1) fprintf(stdout, "Matrix Kernel finished.\n");
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief driver function for computing the matrix operations
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to matrix size structure
 * @param operation - switch-case value for which matrix operation to perform
 *                    1: Matrix addition
 *                    2: Matrix Hadamard product
 *                    3: Sigmoid function
 *                    4: Sigmoid derivative
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 */
void RunMatrixKernel(int argc, char **argv, int &devID, MatrixSize *matrixSize, int operation,
                     float *host_matrixA, float *host_matrixB, float *host_matrixC, float alpha, float beta) {
    // Assign CUDA variables
    hipError_t err;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    int gridX = (int) ceil((float) matrixSize->C_width / (float) threads.x);
    int gridY = (int) ceil((float) matrixSize->C_height / (float) threads.y);
    dim3 grid((unsigned int) gridX, (unsigned int) gridY);

    // Assign computation variables
    float *dev_matrixA = NULL, *dev_matrixB = NULL, *dev_matrixC = NULL;
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Initialize memory on GPU
    MatrixInitCUDA(argc, argv, devID, matrixSize,
                   host_matrixA, host_matrixB, host_matrixC,
                   dev_matrixA, dev_matrixB, dev_matrixC);

    switch (operation) {
        case 1: {
            // Compute Matrix Addition
            MatrixAddKernel<<<grid, threads>>>(dev_matrixA, dev_matrixB, dev_matrixC, alpha, beta,
                    matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) fprintf(stderr, "ERROR in Matrix Add Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 2: {
            // Compute Hadamard Product
            MatrixHadamardKernel<<<grid, threads>>>(dev_matrixA, dev_matrixB, dev_matrixC, alpha, beta,
                    matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Matrix Hadamard Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 3: {
            // Compute Sigmoid function
            MatrixSigmoid<<<grid, threads>>>(dev_matrixA, dev_matrixC, matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Matrix Sigmoid Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 4: {
            // Compute Sigmoid derivative function
            MatrixSigmoidDerivative<<<grid, threads>>>
                                            (dev_matrixA, dev_matrixC, matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess)
                fprintf(stderr, "ERROR in Matrix Sigmoid Derivative Computation: %s\n", hipGetErrorString(err));
            break;
        }
        default: {
            fprintf(stderr, "ERROR: No matrix kernel selected. Operation Aborted");
            break;
        }
    }

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess)
        fprintf(stderr, "ERROR synchronizing Matrix Kernel calculation: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_matrixC, dev_matrixC, matrixC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) fprintf(stderr, "ERROR copying matrix C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_matrixA);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing matrix A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixB);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing matrix B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixC);
    if (err != hipSuccess) fprintf(stderr, "ERROR freeing matrix C on GPU: %s\n", hipGetErrorString(err));
    err = hipDeviceSynchronize();
    if (err != hipSuccess) fprintf(stderr, "ERROR synchronizing Matrix Kernel end: %s\n", hipGetErrorString(err));
    if (LOGGING == 1) fprintf(stdout, "Matrix Kernel finished.\n");

}

//==================================================================================
//=== Test Function
//==================================================================================

void runTest(int argc, char **argv, int devID) {
    int N = 10;
    float *host_A, *host_B, *host_C, *host_D;
    float *host_vA, *host_vB, *host_vC, *host_vD, *host_vE;

    // Create matrices
    MatrixSize *testMatrixSize = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    size_t calcSize = N * N * sizeof(float);
    host_A = (float *) calloc(calcSize, 1);
    host_B = (float *) calloc(calcSize, 1);
    host_C = (float *) calloc(calcSize, 1);
    host_D = (float *) calloc(calcSize, 1);
    SetMatrixSize(testMatrixSize, N, N, N, N, N + 1, N + 1);

    // Create vectors
    VectorSize *testVectorSize = (VectorSize *) calloc(sizeof(VectorSize), 1);
    size_t calcSize_V = N * sizeof(float);
    host_vA = (float *) calloc(calcSize_V, 1);
    host_vB = (float *) calloc(calcSize_V, 1);
    host_vC = (float *) calloc(calcSize_V, 1);
    host_vD = (float *) calloc(calcSize_V, 1);
    host_vE = (float *) calloc(calcSize_V, 1);
    SetVectorSize(testVectorSize, N);

    // Initialize matrix values
    for (int i = 0; i < N * N; i++) {
        host_A[i] = (float) i;
        host_B[i] = (float) i;
    }

    // Initialize vector values
    for (int i = 0; i < N; i++) {
        host_vA[i] = (float) i;
        host_vB[i] = (float) i;
    }

    // MATRIX TESTS

    if (LOGGING == 1) {
        fprintf(stdout, "Matrix A:\n");
        for (int i = 0; i < N + 1; i++) {
            for (int j = 0; j < N + 1; j++) {
                fprintf(stdout, "%6.0f ", host_A[i * j]);
            }
            fprintf(stdout, "\n");
        }

        fprintf(stdout, "\nMatrix B:\n");
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                fprintf(stdout, "%6.0f ", host_B[i * j]);
            }
            fprintf(stdout, "\n");
        }
    }

    RunMatrixKernel(argc, argv, devID, testMatrixSize, 3, host_A, host_B, host_C, 1.0, 1.0);
    RunMatrixKernel(argc, argv, devID, testMatrixSize, 4, host_A, host_B, host_D, 1.0, 1.0);

    if (LOGGING == 1) {
        fprintf(stdout, "\nMatrix C:\n");
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                fprintf(stdout, "%6.10f ", host_C[i * j]);
            }
            fprintf(stdout, "\n");
        }
        fprintf(stdout, "\nMatrix D:\n");
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                fprintf(stdout, "%6.10f ", host_D[i * j]);
            }
            fprintf(stdout, "\n");
        }
    }

    // VECTOR TESTS

    if (LOGGING == 1) {
        fprintf(stdout, "Vector A:\n");
        for (int i = 0; i < N; i++) {
            fprintf(stdout, "%6.0f ", host_vA[i]);
        }
        fprintf(stdout, "\n");
        fprintf(stdout, "\nVector B:\n");

        for (int i = 0; i < N; i++) {
            fprintf(stdout, "%6.0f ", host_vB[i]);
        }
        fprintf(stdout, "\n");
    }

    RunVectorKernel(argc, argv, devID, testVectorSize, 3, host_vA, host_vB, host_vC, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 4, host_vA, host_vB, host_vD, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 5, host_vA, host_vB, host_vE, 1.0, 1.0);

    if (LOGGING == 1) {
        fprintf(stdout, "Vector C:\n");
        for (int i = 0; i < N; i++) {
            fprintf(stdout, "%6.0f ", host_vC[i]);
        }
        fprintf(stdout, "\n");
        fprintf(stdout, "\nVector D:\n");

        for (int i = 0; i < N; i++) {
            fprintf(stdout, "%6.10f ", host_vD[i]);
        }
        fprintf(stdout, "\n");
        fprintf(stdout, "\nVector E:\n");

        for (int i = 0; i < N; i++) {
            fprintf(stdout, "%6.10f ", host_vE[i]);
        }
        fprintf(stdout, "\n");
    }
}

//==================================================================================
//=== Utility Functions
//==================================================================================

void ReadCSV(std::ifstream &file, int elements, float *array)
{
    std::string csvData;
    getline(file, csvData);

    std::istringstream dataStream(csvData);

    for (int col = 0; col < elements; col++){
        std::string value;
        getline(dataStream, value, ',');
        if ( !dataStream.good() )
            break;
        std::istringstream convertor(value);
        convertor >> array[col];
    }
}

void InitializeWeights(float *weights, MatrixSize *dims)
{
    int cols = dims->C_width;
    int rows = dims->C_height;
    int numEl = cols*rows;
    for(int idx = 0; idx < numEl; idx++)
    {
        weights[idx] = ((float) rand() / (RAND_MAX));
    }
}

//==================================================================================
//=== Main Function
//==================================================================================

/**
 * @brief computes weight matrices for a shallow neural network
 * @param argc - from compiler
 * @param argv - from compiler
 * @return 0 if success
 */
int main(int argc, char **argv) {
    // Assign CUDA variables
    int devID = 0;
    hipGetDevice(&devID);
    hipError_t mainErr;
    //runTest(argc, argv, devID);

    // Define NN layer lengths
    unsigned int layer_1 = 784;
    unsigned int layer_2 = 128;
    unsigned int layer_3 = 10;

    // Allocate memory for matrices and vectors
    float *a1, *a2, *a3;    // Activation vectors
    float *z2, *z3;         // Pre-sigmoid intermediary vectors
    float *W1, *W2;         // Weight matrices
    float *y;               // One-hot result vector
    float *del3, *del2;     // Error vectors
    float *scratch1, *scratch2;     // Error vectors
    float *Del2, *Del1;     // Error gradients

    a1 = (float *) calloc((size_t) layer_1, sizeof(float));
    a2 = (float *) calloc((size_t) layer_2, sizeof(float));
    a3 = (float *) calloc((size_t) layer_3, sizeof(float));
    z2 = (float *) calloc((size_t) layer_2, sizeof(float));
    z3 = (float *) calloc((size_t) layer_3, sizeof(float));
    y = (float *) calloc((size_t) layer_3, sizeof(float));
    W1 = (float *) calloc((size_t) layer_2 * layer_1, sizeof(float));
    W2 = (float *) calloc((size_t) layer_3 * layer_2, sizeof(float));
    del3 = (float *) calloc((size_t) layer_3, sizeof(float));
    del2 = (float *) calloc((size_t) layer_2, sizeof(float));
    scratch1 = (float *) calloc((size_t) layer_2, sizeof(float));
    scratch2 = (float *) calloc((size_t) layer_2, sizeof(float));
    Del2 = (float *) calloc((size_t) layer_3 * layer_2, sizeof(float));
    Del1 = (float *) calloc((size_t) layer_2 * layer_1, sizeof(float));

    // Initialize vector and matrix size structures for computation
    MatrixSize *inter2 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *inter3 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *grad1 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *grad2 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *backprop1 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    MatrixSize *backprop2 = (MatrixSize *) calloc(sizeof(MatrixSize), 1);

    VectorSize *activation2 = (VectorSize *) calloc(sizeof(VectorSize), 1);
    VectorSize *activation3 = (VectorSize *) calloc(sizeof(VectorSize), 1);
    VectorSize *delta2 = (VectorSize *) calloc(sizeof(VectorSize), 1);
    VectorSize *delta3 = (VectorSize *) calloc(sizeof(VectorSize), 1);

    if (LOGGING == 1) fprintf(stdout, "Intermediate 2: ");
    SetMatrixSize(inter2, 1, layer_1, layer_2, layer_1, 1, layer_2);
    if (LOGGING == 1) fprintf(stdout, "Intermediate 3: ");
    SetMatrixSize(inter3, 1, layer_2, layer_3, layer_2, 1, layer_3);
    if (LOGGING == 1) fprintf(stdout, "Grad 1: ");
    SetMatrixSize(grad1, 1, layer_2, 1, layer_1, layer_2, layer_1);
    if (LOGGING == 1) fprintf(stdout, "Grad 2: ");
    SetMatrixSize(grad2, 1, layer_3, 1, layer_2, layer_3, layer_2);
    if (LOGGING == 1) fprintf(stdout, "Backprop 1: ");
    SetMatrixSize(backprop1, layer_2, layer_1, layer_2, layer_1, layer_2, layer_1);
    if (LOGGING == 1) fprintf(stdout, "Backprop 2: ");
    SetMatrixSize(backprop2, layer_3, layer_2, layer_3, layer_2, layer_3, layer_2);

    if (LOGGING == 1) fprintf(stdout, "Activation 2: ");
    SetVectorSize(activation2, layer_2);
    if (LOGGING == 1) fprintf(stdout, "Activation 3: ");
    SetVectorSize(activation3, layer_3);
    if (LOGGING == 1) fprintf(stdout, "Delta 2: ");
    SetVectorSize(delta2, layer_2);
    if (LOGGING == 1) fprintf(stdout, "Delta 3: ");
    SetVectorSize(delta3, layer_3);

    // Set number of epochs and samples
    int epochs = 1; // Number of training epochs (iterations through data)
    int num_train = 20000;   // Number of samples;
    int num_test = 5000;

    // Initalize weights
    InitializeWeights(W1, grad1);
    InitializeWeights(W2, grad2);

    //Perform neural network training
    for (int epoch = 0; epoch < epochs; epoch++) {

        // Open training data files
        std::ifstream x_train_data("./data/train_img.csv");
        std::ifstream y_train_data("./data/train_res.csv");

        for (int sample = 0; sample < num_train; sample++) {
            // LOAD a1 AND y VECTORS:
            ReadCSV(x_train_data, layer_1, a1);
            ReadCSV(y_train_data, layer_3, y);

            // FORWARD PROPOGATION:
            MatrixMultiplyCUBLAS(argc, argv, devID, inter2, a1, W1, z2, 1.0, 1.0, false, true); // Compute z2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "z2 Computation: %s\n", hipGetErrorString(mainErr));
            RunVectorKernel(argc, argv, devID, activation2, 4, z2, z2, a2, 1.0, 1.0);           // Compute a2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "a2 Computation: %s\n", hipGetErrorString(mainErr));
            MatrixMultiplyCUBLAS(argc, argv, devID, inter3, a2, W2, z3, 1.0, 1.0, false, true); // Compute z3
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "z3 Computation: %s\n", hipGetErrorString(mainErr));
            RunVectorKernel(argc, argv, devID, activation3, 4, z3, z3, a3, 1.0, 1.0);           // Compute a3
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "a3 Computation: %s\n", hipGetErrorString(mainErr));

            // BACKWARD PROPOGATION:
            RunVectorKernel(argc, argv, devID, delta3, 1, z3, y, del3, 1.0, (float) -1.0);           // Compute del3
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "del3 Computation: %s\n", hipGetErrorString(mainErr));

            MatrixMultiplyCUBLAS(argc, argv, devID, inter3, del3, W2, scratch1, 1.0, 1.0, false, false); // Compute del2 lhs
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "pre-del2 lhs Computation: %s\n", hipGetErrorString(mainErr));

            RunVectorKernel(argc, argv, devID, delta2, 5, z2, y, scratch2, 1.0, (float) -1.0);           // Compute del2 rhs
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "pre-del2 rhs Computation: %s\n", hipGetErrorString(mainErr));

            RunVectorKernel(argc, argv, devID, delta2, 2, scratch1, scratch2, del2, 1.0, (float) -1.0);  // Compute del2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "del2 Computation: %s\n", hipGetErrorString(mainErr));

            MatrixMultiplyCUBLAS(argc, argv, devID, grad1, del2, a1, Del1, 1.0, 1.0, true, false);   // Compute Del1
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "Del1 Computation: %s\n", hipGetErrorString(mainErr));

            MatrixMultiplyCUBLAS(argc, argv, devID, grad2, del3, a2, Del2, 1.0, 1.0, true, false);   // Compute Del2
            mainErr = hipGetLastError();
            if (mainErr != hipSuccess) fprintf(stderr, "Del2 Computation: %s\n", hipGetErrorString(mainErr));

            // Gradient descent
            RunMatrixKernel(argc, argv, devID, backprop1, 1, W1, Del1, W1, 1.0,
                            (float) -1.0 / (float) num_train); // Compute new W1
            RunMatrixKernel(argc, argv, devID, backprop2, 1, W2, Del2, W2, 1.0,
                            (float) -1.0 / (float) num_train); // Compute new W2
            hipDeviceSynchronize();
            if( (sample % 1000) == 0) printf("Iteration: %d\n", sample);
            if( (sample % 5000) == 0) hipDeviceReset();
        }
        // Close training data files
        x_train_data.close();
        y_train_data.close();

    }

    // Open verification data files
    std::ifstream x_test_data("./data/tests_img.csv");
    std::ifstream y_test_data("./data/tests_res.csv");

    int correct = 0;

    for(int test_sample = 0; test_sample < num_test; test_sample++)
    {
        // LOAD a1 AND y VECTORS:
        ReadCSV(x_test_data, layer_1, a1);
        ReadCSV(y_test_data, layer_3, y);

        // FORWARD PROPOGATION:
        MatrixMultiplyCUBLAS(argc, argv, devID, inter2, a1, W1, z2, 1.0, 1.0, false, true); // Compute z2
        mainErr = hipGetLastError();
        if (mainErr != hipSuccess) fprintf(stderr, "z2 Computation: %s\n", hipGetErrorString(mainErr));
        RunVectorKernel(argc, argv, devID, activation2, 4, z2, z2, a2, 1.0, 1.0);           // Compute a2
        mainErr = hipGetLastError();
        if (mainErr != hipSuccess) fprintf(stderr, "a2 Computation: %s\n", hipGetErrorString(mainErr));
        MatrixMultiplyCUBLAS(argc, argv, devID, inter3, a2, W2, z3, 1.0, 1.0, false, true); // Compute z3
        mainErr = hipGetLastError();
        if (mainErr != hipSuccess) fprintf(stderr, "z3 Computation: %s\n", hipGetErrorString(mainErr));
        RunVectorKernel(argc, argv, devID, activation3, 4, z3, z3, a3, 1.0, 1.0);           // Compute a3
        mainErr = hipGetLastError();
        if (mainErr != hipSuccess) fprintf(stderr, "a3 Computation: %s\n", hipGetErrorString(mainErr));

        float a3max = 0.0;
        int a3max_idx = 0;
        int ymax_idx = 0;

        for(int i = 0; i < layer_3; i++)
        {
            if(a3[i] > a3max)
            {
                a3max = a3[i];
                a3max_idx = i;
            }
            if(y[i] == 1) ymax_idx = i;
        }
        if(ymax_idx == a3max_idx) correct++;
    }
    x_test_data.close();
    y_test_data.close();
    printf("The network correctly identified %d of %d samples\n", correct, num_test);
    return 0;
}
